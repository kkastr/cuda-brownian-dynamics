#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

__global__ void rng_setup_kernel(unsigned int seed,hiprandStatePhilox4_32_10_t *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(seed, id, 0, &state[id]);
}


__device__ void integration_kernel(float dt, int steps, float *x, float *y, float *z,hiprandStatePhilox4_32_10_t *state)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  float xrn;
  float yrn;
  float zrn;
  float updf;
  float prob;

  hiprandStatePhilox4_32_10_t localState = state[i];

  xrn = hiprand_uniform(&localState)-0.5;
  yrn = hiprand_uniform(&localState)-0.5;
  zrn = hiprand_uniform(&localState)-0.5;

  state[i] = localState;

  x[i] = x[i] + prf*xrn*dt;
  y[i] = y[i] + prf*yrn*dt;
  z[i] = z[i] + prf*zrn*dt;

  float r = sqrt(x[i]*x[i] + y[i]*y[i]);

  //TODO: fix boundary conditions
  if (r>rcav){
    x[i] = x[i]*rcav/r;
    y[i] = y[i]*rcav/r;
  }
  if( z[i] < zmin){
    z[i] = zmin;
  }
  if( z[i] > lcav){
    z[i] = lcav-(float)1;
  }
}


int main(int argc, char* argv[])
{
  unsigned int seed;
  double tmax;

  int kernel_typeflag;

  if (argc > 1){
		seed = atoi(argv[0]); //sim seed
    tmax = atof(argv[1]); //maximum runtime in timesteps;
	}
	else {
		printf("No arguments given. \n Need to provide the following: seed, tmax.\n");
		return 1;
	}

  srand(seed);
  double t = 0;
  int steps = 0;
  float dt = 0.01;
  int outputfreq = 1000;
  float kT = 1, m = 1, gamma = 1;


  float prf = sqrt((2*kT*gamma)/(m*dt));
  float *hx, *hy, *hz, *d_x, *d_y, *d_z;

  char cout_pos[64];
	sprintf(cout_pos,"trajectory.xyz";

  FILE *cout_position;
  cout_position=fopen(cout_pos,"w");

  const unsigned int threadsPerBlock = 64;
  const unsigned int blockCount = 64;
  int N = threadsPerBlock * blockCount;

  hx = (float*)malloc(N*sizeof(float));
  hy = (float*)malloc(N*sizeof(float));
  hz = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));
  hipMalloc(&d_z, N*sizeof(float));

  hiprandStatePhilox4_32_10_t *devPHILOXStates;

  hipMalloc((void **)&devPHILOXStates, N*sizeof(hiprandStatePhilox4_32_10_t));

  rng_setup_kernel<<<blockCount, threadsPerBlock>>>(seed,devPHILOXStates);

  for (int j=0;j<N;j++){
    hx[j] = 0.0f;
    hy[j] = 0.0f;
    hz[j] = 0.0f;

  }

  hipMemcpy(d_x, hx, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, hy, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, hz, N*sizeof(float), hipMemcpyHostToDevice);

  while (t < tmax)
  {
    //TODO: fix inputs
    integration_kernel<<<blockCount, threadsPerBlock>>>(kernel_typeflag, dt, steps, d_x, d_y, d_z, devPHILOXStates);


    if (steps%outputfreq==0)
    {
        hipMemcpy(hx, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(hy, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(hz, d_z, N*sizeof(float), hipMemcpyDeviceToHost);

      fprintf(cout_position, "%i", N)
      fprintf(cout_position, "comment")
      for (int i = 0; i < N; i++)
      {
        fprintf(cout_position,"%i,%f,%f,%f\n",i,hx[i],hy[i],hz[i]);

      }
    }

    t+=dt;
    steps++;
  }
  hipFree(devPHILOXStates);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  free(hx);
  free(hy);
  free(hz);
}
