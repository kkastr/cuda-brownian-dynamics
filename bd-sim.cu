#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

__global__ void rng_setup_kernel(unsigned int seed,hiprandStatePhilox4_32_10_t *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(seed, id, 0, &state[id]);
}


__device__ void integration_kernel(float dt, float prf, float3 lbox, float *x, float *y, float *z,hiprandStatePhilox4_32_10_t *state)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  float xrn;
  float yrn;
  float zrn;

  hiprandStatePhilox4_32_10_t localState = state[i];

  xrn = hiprand_uniform(&localState)-0.5;
  yrn = hiprand_uniform(&localState)-0.5;
  zrn = hiprand_uniform(&localState)-0.5;

  state[i] = localState;

  x[i] = x[i] + prf*xrn*dt;
  y[i] = y[i] + prf*yrn*dt;
  z[i] = z[i] + prf*zrn*dt;

  float r = sqrt(x[i]*x[i] + y[i]*y[i]);

  //TODO: use better boundary conditions
  if (abs(x) > lbox.x/2){
    x[i] = x[i];
  }
  if (abs(y > lbox.y/2)){
    y[i] = y[i];
  }
  if (abs(z > lbox.z/2)){
    z[i] = z[i];
  }
}


int main(int argc, char* argv[])
{
  unsigned int seed;
  double tmax;

  int kernel_typeflag;

  if (argc > 1){
		seed = atoi(argv[0]); //sim seed
    tmax = atof(argv[1]); //maximum runtime in timesteps;
	}
	else {
		printf("No arguments given. \n Need to provide the following: seed, tmax.\n");
		return 1;
	}

  srand(seed);
  double t = 0;
  int steps = 0;
  float dt = 0.01;
  int outputfreq = 1000;
  float kT = 1, m = 1, gamma = 1;
  float3 lbox;

  lbox.x = lbox.y = lbox.z = 25;


  float prf = sqrt((2*kT*gamma)/(m*dt));
  float *hx, *hy, *hz, *d_x, *d_y, *d_z;

  char cout_pos[64];
	sprintf(cout_pos,"trajectory.xyz");

  FILE *cout_position;
  cout_position=fopen(cout_pos,"w");

  const unsigned int threadsPerBlock = 64;
  const unsigned int blockCount = 64;
  int N = threadsPerBlock * blockCount;

  hx = (float*)malloc(N*sizeof(float));
  hy = (float*)malloc(N*sizeof(float));
  hz = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));
  hipMalloc(&d_z, N*sizeof(float));

  hiprandStatePhilox4_32_10_t *devPHILOXStates;

  hipMalloc((void **)&devPHILOXStates, N*sizeof(hiprandStatePhilox4_32_10_t));

  rng_setup_kernel<<<blockCount, threadsPerBlock>>>(seed,devPHILOXStates);

  for (int j=0;j<N;j++){
    hx[j] = 0.0f;
    hy[j] = 0.0f;
    hz[j] = 0.0f;

  }

  hipMemcpy(d_x, hx, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, hy, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, hz, N*sizeof(float), hipMemcpyHostToDevice);

  while (t < tmax)
  {

    integration_kernel<<<blockCount, threadsPerBlock>>>(dt, prf, lbox, d_x, d_y, d_z, devPHILOXStates);


    if (steps%outputfreq==0)
    {
        hipMemcpy(hx, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(hy, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(hz, d_z, N*sizeof(float), hipMemcpyDeviceToHost);

      fprintf(cout_position, "%i", N);
      fprintf(cout_position, "comment");
      for (int i = 0; i < N; i++)
      {
        fprintf(cout_position,"%i,%f,%f,%f\n",i,hx[i],hy[i],hz[i]);

      }
    }

    t+=dt;
    steps++;
  }
  hipFree(devPHILOXStates);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  free(hx);
  free(hy);
  free(hz);
}
